
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define uchar unsigned char
#define uint unsigned int

__global__ void computeBoardNextState(uchar* board, uchar* temp, uint m, uint n){
    
}

__global__ void copyBoard(uchar* board, uchar* temp, uint m, uint n){ // is this the most efficient copy?
   
}

inline void readBoard(uchar* h_board, uint m, uint n){
    char t;
    for(int i = 1; i < m - 1; ++i){
        for(int j = 1; j < n - 1; ++j){
            scanf("%c", &t);
            h_board[i * n + j] = (t == '#');
        }
        scanf("%c", &t); // newline
    }
}

inline void printBoard(uchar* h_board, uint m, uint n){
    for(int i = 1; i < m - 1; ++i){
        for(int j = 1; j < n - 1; ++j){
            printf("%c", (h_board[i * n + j] ? '#' : '.'));
        }
        printf("\n");
    }
}

int main(){
    
}