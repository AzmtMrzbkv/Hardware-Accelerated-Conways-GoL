
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void computeNextState(char* grid, int m, int n){
    int i = blockIdx.x, j = threadIdx.x, k = 0;

    // left - right
    if(j > 0 && grid[i * n + j - 1])
        ++k;
    if(j < n - 1 && grid[i * n + j + 1])
        ++k;

    // top
    if(i > 0){
        if(grid[(i - 1) * n + j])
            ++k;
        if(j > 0 && grid[(i - 1) * n + j - 1])
            ++k;
        if(j < n - 1 && grid[(i - 1) * n + j + 1])
            ++k;
    }

    // bottom
    if(i < m - 1){
        if(grid[(i + 1) * n + j])
            ++k;
        if(j > 0 && grid[(i + 1) * n + j - 1])
            ++k;
        if(j < n - 1 && grid[(i + 1) * n + j + 1])
            ++k;
    }

    __syncthreads();

    // define new state
    if(k == 2)
        grid[i * n + j] = grid[i * n + j];
    else if(k == 3)
        grid[i * n + j] = 1;
    else
        grid[i * n + j] = 0;

    __syncthreads();
}

int main(){
    int m, n, g;
    char *h_grid, *d_grid, t;

    scanf("%d %d %d ", &m, &n, &g);
    g = n;
    n = m;
    h_grid = (char*)malloc(sizeof(char) * m * n);

    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            scanf("%c", &t);
            h_grid[i * n + j] = (t == '#');
        }
        scanf("%c", &t); // newline
    }

    hipMalloc((void**)&d_grid, sizeof(char) * m * n);
    hipMemcpy(d_grid, h_grid, sizeof(char) * m * n, hipMemcpyHostToDevice);
  
    for(; g > 0; --g)
        computeNextState<<<m, n>>>(d_grid, m, n); // implicit synchronization 

    hipMemcpy(h_grid, d_grid, sizeof(char) * m * n, hipMemcpyDeviceToHost);

    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            printf("%c", (h_grid[i * n + j] ? '#' : '.'));
        }
        printf("\n");
    }

    hipFree(d_grid);
    free(h_grid);
}