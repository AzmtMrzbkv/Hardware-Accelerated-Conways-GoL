
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define uchar unsigned char
#define uint unsigned int

__global__ void computeBoardNextState(uchar* board, uchar* temp, uint m, uint n){
    uint idx = (uint)blockDim.x * blockIdx.x + threadIdx.x; 
    if(idx >= (m - 2) * (n - 2)) 
        return;

    uint i = idx / (n - 2), j = idx % (n - 2), k = 0;

    // left - right
    k += board[(i + 1) * n + (j + 1) - 1];
    k += board[(i + 1) * n + (j + 1) + 1];

    k += board[(i + 1 - 1) * n + (j + 1)];
    k += board[(i + 1 - 1) * n + (j + 1) - 1];
    k += board[(i + 1 - 1) * n + (j + 1) + 1];

    k += board[(i + 1 + 1) * n + (j + 1)];
    k += board[(i + 1 + 1) * n + (j + 1) - 1];
    k += board[(i + 1 + 1) * n + (j + 1) + 1];

    if(k == 2)
        temp[(i + 1) * n + (j + 1)] = board[(i + 1) * n + (j + 1)];
    else
        temp[(i + 1) * n + (j + 1)] = (k == 3);
}

__global__ void copyBoard(uchar* board, uchar* temp, uint m, uint n){ // is this the most efficient copy?
    uint idx = (uint)blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= (m - 2) * (n - 2)) 
        return;

    uint i = idx / (n - 2), j = idx % (n - 2);
    board[(i + 1) * n + (j + 1)] = temp[(i + 1) * n + (j + 1)];
}

inline void readBoard(uchar* h_board, uint m, uint n){
    char t;
    for(int i = 1; i < m - 1; ++i){
        for(int j = 1; j < n - 1; ++j){
            scanf("%c", &t);
            h_board[i * n + j] = (t == '#');
        }
        scanf("%c", &t); // newline
    }
}

inline void printBoard(uchar* h_board, uint m, uint n){
    for(int i = 1; i < m - 1; ++i){
        for(int j = 1; j < n - 1; ++j){
            printf("%c", (h_board[i * n + j] ? '#' : '.'));
        }
        printf("\n");
    }
}

int main(){
    uint m, n, g, boardRows, boardCols;
    uchar *h_board, *d_res_board, *d_temp_board;

    scanf("%u %u %u ", &m, &n, &g);

    // add paddings to remove out-of-borders checks
    boardRows = m + 2;
    boardCols = n + 2;

    h_board = (uchar*)malloc(sizeof(uchar) * boardRows * boardCols);
    memset((void*)h_board, 0, sizeof(uchar) * boardRows * boardCols);
    readBoard(h_board, boardRows, boardCols);

    hipMalloc((void**)&d_res_board, sizeof(uchar) * boardRows * boardCols);
    hipMalloc((void**)&d_temp_board, sizeof(uchar) * boardRows * boardCols);

    hipMemcpy(d_res_board, h_board, sizeof(uchar) * boardRows * boardCols, hipMemcpyHostToDevice);
    hipMemset((void*)d_temp_board, 0, sizeof(uchar) * boardRows * boardCols); // allocated memory is not cleared
  
    const uint threadNum = 512;
    const uint blockNum = ceil((double)m * n / threadNum); // set these numbers for now. ideally use dim3

    for(int i = 0; i < g; ++i){
        computeBoardNextState<<<blockNum, threadNum>>>(d_res_board, d_temp_board, boardRows, boardCols); // implicit synchronization before next kernel
        copyBoard<<<blockNum, threadNum>>>(d_res_board, d_temp_board, boardRows, boardCols); // copy from temp board to res board
    }

    hipMemcpy(h_board, d_res_board, sizeof(uchar) * boardRows * boardCols, hipMemcpyDeviceToHost);
    printBoard(h_board, boardRows, boardCols);

    hipFree(d_res_board);
    hipFree(d_temp_board);
    free(h_board);
}